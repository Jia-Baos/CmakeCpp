#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(void) { printf("Hello World from GPU!\n"); }

__global__ void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) y[i] = x[i] + y[i];
}

int main() {
  printf("Hello World from CPU!\n");

  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Number of SM: %d\n", prop.multiProcessorCount);
    printf("  Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Maximum threads per sm: %d\n", prop.maxBlocksPerMultiProcessor);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  }

  cuda_hello<<<1, 10>>>();
  hipDeviceReset();

  // int N = 1 << 20;
  // float *x, *y;

  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N * sizeof(float));
  // hipMallocManaged(&y, N * sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // // Run kernel on 1M elements on the GPU
  // int blockSize = 256;
  // int numBlocks = (N + blockSize - 1) / blockSize;
  // add<<<numBlocks, blockSize>>>(N, x, y);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);

  return 0;
}
